#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////
//  SolverChannels.cpp
//  Implementation of the Class SolverChannels
//  Created on:      27-Dec-2013 7:57:50 PM
//  Original author: Saeed Shariati
///////////////////////////////////////////////////////////

#include "SolverChannels.h"

#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <math.h>

using namespace pn2s::models;

#define SINGULARITY 1.0e-6

//A mask to check INSTANT variable in the channel
#define INSTANT_X 1
#define INSTANT_Y 2
#define INSTANT_Z 4

#define NUMBER_OF_MULTI_PROCESSOR 8

SolverChannels::SolverChannels(): _stream(0)
{
}

SolverChannels::~SolverChannels()
{
}

void SolverChannels::AllocateMemory(models::ModelStatistic& s, hipStream_t stream)
{
	_m_statistic = s;
	_stream = stream;

	if(_m_statistic.nChannels_all == 0)
		return;

	_state.AllocateMemory(_m_statistic.nChannels_all*3);
	_comptIndex.AllocateMemory(_m_statistic.nChannels_all);
	_channel_base.AllocateMemory(_m_statistic.nChannels_all);
	_channel_currents.AllocateMemory(_m_statistic.nChannels_all);

	int threadSize = min(max((int)_m_statistic.nChannels_all/NUMBER_OF_MULTI_PROCESSOR,16), 32);
	_threads=dim3(2,threadSize, 1);
	_blocks=dim3(max((int)(ceil(_m_statistic.nChannels_all / _threads.y)),1), 1);
}

void SolverChannels::PrepareSolver(PField<TYPE_, ARCH_>*  Vm)
{
	if(_m_statistic.nChannels_all)
	{
		_state.Host2Device_Async(_stream);
		_channel_base.Host2Device_Async(_stream);
		_channel_currents.Host2Device_Async(_stream);
		_comptIndex.Host2Device_Async(_stream);
		_Vm = Vm;

		int threadSize = min(max((int)_m_statistic.nChannels_all/8,16), 32);
		_threads=dim3(2,threadSize, 1);
		_blocks=dim3(max((int)(ceil(_m_statistic.nChannels_all / _threads.y)),1), 1);
	}
}

/**
 * KERNELS
 */
extern __shared__ TYPE_ shmem[];

__global__ void advanceChannels(
		TYPE_* v,
		int* compIndex,
		TYPE_* state,
		pn2s::models::ChannelType* ch,
		pn2s::models::ChannelCurrent* current,
		size_t size, TYPE_ dt)
{
	TYPE_ temp, temp2, A, B;
	TYPE_* data = shmem;
	int idx = blockIdx.x * blockDim.y + threadIdx.y;
	if (idx >= size)
		return;

	int i = threadIdx.y * 2 + threadIdx.x;
	data[i] = 1.0;

	TYPE_ power = ch[idx]._xyz_power[threadIdx.x];
	if ( power > 0.0 )
	{
		int cIdx = compIndex[idx];
		TYPE_ x = v[cIdx];


		temp = ch[idx]._xyz_params[threadIdx.x][PARAMS_MIN];
		temp2 = ch[idx]._xyz_params[threadIdx.x][PARAMS_MAX];
		// Check boundaries
		x = fmax(temp, x);
		x = fmin(temp2, x);

		// Calculate new states
		TYPE_ dx = ( temp2 - temp ) / ch[idx]._xyz_params[threadIdx.x][PARAMS_DIV];

		if ( fabs(ch[idx]._xyz_params[threadIdx.x][PARAMS_A_F]) < SINGULARITY ) {
			temp = 0.0;
			A = 0.0;
		} else {
			temp2 = ch[idx]._xyz_params[threadIdx.x][PARAMS_A_C] + exp( ( x + ch[idx]._xyz_params[threadIdx.x][PARAMS_A_D] ) / ch[idx]._xyz_params[threadIdx.x][PARAMS_A_F] );
			if ( fabs( temp2 ) < SINGULARITY ) {
				temp2 = ch[idx]._xyz_params[threadIdx.x][PARAMS_A_C] + exp( ( x + dx/10.0 + ch[idx]._xyz_params[threadIdx.x][PARAMS_A_D] ) / ch[idx]._xyz_params[threadIdx.x][PARAMS_A_F] );
				temp = ( ch[idx]._xyz_params[threadIdx.x][PARAMS_A_A] + ch[idx]._xyz_params[threadIdx.x][PARAMS_A_B] * (x + dx/10 ) ) / temp2;

				temp2 = ch[idx]._xyz_params[threadIdx.x][PARAMS_A_C] + exp( ( x - dx/10.0 + ch[idx]._xyz_params[threadIdx.x][PARAMS_A_D] ) / ch[idx]._xyz_params[threadIdx.x][PARAMS_A_F] );
				temp += ( ch[idx]._xyz_params[threadIdx.x][PARAMS_A_A] + ch[idx]._xyz_params[threadIdx.x][1] * (x - dx/10 ) ) / temp2;
				temp /= 2.0;

				A = temp;
			} else {
				temp = ( ch[idx]._xyz_params[threadIdx.x][PARAMS_A_A] + ch[idx]._xyz_params[threadIdx.x][PARAMS_A_B] * x) / temp2;
				A = temp;
			}
		}

		if ( fabs( ch[idx]._xyz_params[threadIdx.x][9] ) < SINGULARITY ) {
			B = 0.0;
		} else {
			temp2 = ch[idx]._xyz_params[threadIdx.x][7] + exp( ( x + ch[idx]._xyz_params[threadIdx.x][8] ) / ch[idx]._xyz_params[threadIdx.x][9] );
			if ( fabs( temp2 ) < SINGULARITY ) {
				temp2 = ch[idx]._xyz_params[threadIdx.x][7] + exp( ( x + dx/10.0 + ch[idx]._xyz_params[threadIdx.x][8] ) / ch[idx]._xyz_params[threadIdx.x][9] );
				temp = (ch[idx]._xyz_params[threadIdx.x][5] + ch[idx]._xyz_params[threadIdx.x][6] * (x + dx/10) ) / temp2;
				temp2 = ch[idx]._xyz_params[threadIdx.x][7] + exp( ( x - dx/10.0 + ch[idx]._xyz_params[threadIdx.x][8] ) / ch[idx]._xyz_params[threadIdx.x][9] );
				temp += (ch[idx]._xyz_params[threadIdx.x][5] + ch[idx]._xyz_params[threadIdx.x][6] * (x - dx/10) ) / temp2;
				temp /= 2.0;
				B = temp;
			} else {
				B = (ch[idx]._xyz_params[threadIdx.x][5] + ch[idx]._xyz_params[threadIdx.x][6] * x ) / temp2;
			}
		}

		if ( fabs( temp2 ) > SINGULARITY )
			B += temp;

		temp2 = state[3*idx+threadIdx.x];
		if ( ch[idx]._instant& INSTANT_X )
			state[3*idx+threadIdx.x] = A / B;
		else
		{
			temp = 1.0 + dt / 2.0 * B; //new value for temp
			state[3*idx+threadIdx.x] = ( temp2 * ( 2.0 - temp ) + dt * A ) / temp;
		}

		//Update channels characteristics
		data[i] = temp2;
		if (power > 1)
		{
			data[i] *= temp2;
			if (power > 2)
			{
				data[i] *= temp2;
				if (power > 3)
				{
					data[i] *= temp2;
					if (power > 4)
					{
						data[i] = pow( temp2, power);
					}
				}
			}
		}
		__syncthreads();
		if(!threadIdx.x)
			current[idx]._gk = ch[idx]._gbar * data[i] * data[i+1];
	}
}

void SolverChannels::Input()
{

}

void SolverChannels::Process()
{
	if(_m_statistic.nChannels_all < 1)
		return;
	int smem_size = (sizeof(TYPE_) * _threads.x * _threads.y);
//	_Vm->Device2Host();
//	_Vm->print();
//	_state.Device2Host();
//	_state.print();

	advanceChannels <<<_blocks, _threads,smem_size, _stream>>> (
			_Vm->device,
			_comptIndex.device,
			_state.device,
			_channel_base.device,
			_channel_currents.device,
			_m_statistic.nChannels_all,
			_m_statistic.dt);

//	_state.Device2Host();
//	_state.print();
//
//	_channel_currents.Device2Host();
//	_channel_currents.print();
	assert(hipSuccess == hipGetLastError());
}

void SolverChannels::Output()
{
	_channel_currents.Device2Host_Async(_stream);
}

/**
 * Set/Get methods
 */

void SolverChannels::SetGateXParams(int index, vector<double>& params)
{
	for (int i = 0; i < min((int)params.size(),13); ++i)
		_channel_base[index]._xyz_params[0][i] = (TYPE_)params[i];
}
void SolverChannels::SetGateYParams(int index, vector<double>& params)
{
	for (int i = 0; i < min((int)params.size(),13); ++i)
		_channel_base[index]._xyz_params[1][i] = (TYPE_)params[i];
}
void SolverChannels::SetGateZParams(int index, vector<double>& params)
{
	for (int i = 0; i < min((int)params.size(),13); ++i)
		_channel_base[index]._xyz_params[2][i] = (TYPE_)params[i];
}

void SolverChannels::SetValue(int index, FIELD::TYPE field, TYPE_ value)
{
	switch(field)
	{
		case FIELD::CH_GBAR:
			_channel_base[index]._gbar = value;
			break;
		case FIELD::CH_GK:
			_channel_currents[index]._gk = value;
			break;
		case FIELD::CH_EK:
			_channel_currents[index]._ek = value;
			break;
		case FIELD::CH_X_POWER:
			_channel_base[index]._xyz_power[0] = (unsigned char)value;
			break;
		case FIELD::CH_Y_POWER:
			_channel_base[index]._xyz_power[1] = (unsigned char)value;
			break;
		case FIELD::CH_Z_POWER:
			_channel_base[index]._xyz_power[2] = (unsigned char)value;
			break;
		case FIELD::CH_X:
			_state[3*index] = value;
			break;
		case FIELD::CH_Y:
			_state[3*index+1] = value;
			break;
		case FIELD::CH_Z:
			_state[3*index+2] = value;
			break;
		case FIELD::CH_COMPONENT_INDEX:
			_comptIndex[index] = (int)value;
			break;
	}
}

TYPE_ SolverChannels::GetValue(int index, FIELD::TYPE field)
{
//	switch(field)
//	{
//		case FIELD::CH_GBAR:
//			return _gbar[index];
//		case FIELD::CH_X_POWER:
//			return _xPower[index];
//		case FIELD::CH_Y_POWER:
//			return _yPower[index];
//		case FIELD::CH_Z_POWER:
//			return _zPower[index];
//	}
	return 0;
}
