#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////
//  SolverGates.cpp
//  Implementation of the Class SolverGates
//  Created on:      27-Dec-2013 7:57:50 PM
//  Original author: Saeed Shariati
///////////////////////////////////////////////////////////

#include "SolverGates.h"

#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <math.h>

using namespace pn2s::models;

#define SINGULARITY 1.0e-6

//A mask to check INSTANT variable in the channel
#define INSTANT_X 1
#define INSTANT_Y 2
#define INSTANT_Z 4
#define IS_SECOND_GATE 8

#define NUMBER_OF_MULTI_PROCESSOR 8

#define PARAM_SIZE	13

SolverGates::SolverGates(): _stream(0), _Vm(0)
{
}

SolverGates::~SolverGates()
{
}

void SolverGates::AllocateMemory(models::ModelStatistic& s, hipStream_t stream)
{
	_m_statistic = s;
	_stream = stream;

	if(_m_statistic.nGates <= 0)
		return;

	_ch_currents_gk_ek.AllocateMemory(_m_statistic.nChannels_all);//TODO: remove

	_state.AllocateMemory(_m_statistic.nGates, 0);
	_gk.AllocateMemory(_m_statistic.nChannels_all, 0); //Channel currents

	//Indices
	_comptIndex.AllocateMemory(_m_statistic.nGates, 0);
	_channelIndex.AllocateMemory(_m_statistic.nGates, 0);
	_gateIndex.AllocateMemory(_m_statistic.nGates, 0);

	//Constant values
	_ek.AllocateMemory(_m_statistic.nGates, 0);
	_gbar.AllocateMemory(_m_statistic.nGates, 0);
	_power.AllocateMemory(_m_statistic.nGates, 0);
	_params.AllocateMemory(_m_statistic.nGates, 0);
	_params_div_min_max.AllocateMemory(_m_statistic.nGates, 0);

	int threadSize = min(max((int)_m_statistic.nChannels_all/8,16), 32);
	_threads=dim3(2,threadSize, 1);
	_blocks=dim3(max((int)(ceil((double)_m_statistic.nChannels_all / _threads.y)),1), 1);
}

void SolverGates::PrepareSolver(PField<TYPE_>*  Vm)
{
	if(_m_statistic.nGates)
	{
		_ch_currents_gk_ek.Host2Device_Async(_stream);

		_state.Host2Device_Async(_stream);
		_gk.Host2Device_Async(_stream);
		_comptIndex.Host2Device_Async(_stream);
		_channelIndex.Host2Device_Async(_stream);
		_gateIndex.Host2Device_Async(_stream);
		_ek.Host2Device_Async(_stream);
		_gbar.Host2Device_Async(_stream);
		_power.Host2Device_Async(_stream);
		_params.Host2Device_Async(_stream);
		_params_div_min_max.Host2Device_Async(_stream);
		_Vm = Vm;

		_threads=dim3(32);
		_blocks=dim3(ceil(_m_statistic.nGates / (double)_threads.x));
	}
}

/**
 * KERNELS
 */
__global__ void advanceGates(
		TYPE_*  state,
		TYPE_*  gk,
		TYPE2_* current,
		TYPE_*  power,
		pn2s::models::GateParams* params,
		TYPE3_* div_min_max,
		TYPE_* gbar,
		TYPE_*  ek,
		int*  comptIndex,
		int*  channelIndex,
		int*  gateIndex,
		TYPE_* v,
		size_t size, TYPE_ dt)
{
	extern __shared__ TYPE2_ data[];
	TYPE_ temp, temp2, A, B;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= size)
		return;

	int ch_idx = channelIndex[idx];
	int fi = gateIndex[idx];

	if ( power[idx] > 0.0 )
	{
		TYPE_ x = v[comptIndex[idx]];

		temp = div_min_max[idx].y;
		temp2 = div_min_max[idx].z;

		// Calculate new states
		TYPE_ dx = ( temp2 - temp ) / div_min_max[idx].x;

		// Check boundaries
		x = fmax(temp, x);
		x = fmin(temp2, x);


		if ( fabs(params[idx].p[PARAMS_A_F]) < SINGULARITY ) {
			temp = 0.0;
			A = 0.0;
		} else {
			temp2 = params[idx].p[PARAMS_A_C] + exp( ( x + params[idx].p[PARAMS_A_D] ) / params[idx].p[PARAMS_A_F] );
			if ( fabs( temp2 ) < SINGULARITY ) {
				temp2 = params[idx].p[PARAMS_A_C] + exp( ( x + dx/10.0 + params[idx].p[PARAMS_A_D] ) / params[idx].p[PARAMS_A_F] );
				temp = ( params[idx].p[PARAMS_A_A] + params[idx].p[PARAMS_A_B] * (x + dx/10 ) ) / temp2;

				temp2 = params[idx].p[PARAMS_A_C] + exp( ( x - dx/10.0 + params[idx].p[PARAMS_A_D] ) / params[idx].p[PARAMS_A_F] );
				temp += ( params[idx].p[PARAMS_A_A] + params[idx].p[1] * (x - dx/10 ) ) / temp2;
				temp /= 2.0;

				A = temp;
			} else {
				temp = ( params[idx].p[PARAMS_A_A] + params[idx].p[PARAMS_A_B] * x) / temp2;
				A = temp;
			}
		}

		if ( fabs( params[idx].p[PARAMS_B_F] ) < SINGULARITY ) {
			B = 0.0;
		} else {
			temp2 = params[idx].p[7] + exp( ( x + params[idx].p[8] ) / params[idx].p[9] );
			if ( fabs( temp2 ) < SINGULARITY ) {
				temp2 = params[idx].p[7] + exp( ( x + dx/10.0 + params[idx].p[8] ) / params[idx].p[9] );
				temp = (params[idx].p[5] + params[idx].p[6] * (x + dx/10) ) / temp2;
				temp2 = params[idx].p[7] + exp( ( x - dx/10.0 + params[idx].p[8] ) / params[idx].p[9] );
				temp += (params[idx].p[5] + params[idx].p[6] * (x - dx/10) ) / temp2;
				temp /= 2.0;
				B = temp;
			} else {
				B = (params[idx].p[5] + params[idx].p[6] * x ) / temp2;
			}
		}

		if ( fabs( temp2 ) > SINGULARITY )
			B += temp;

		temp2 = state[idx];
		temp = 1.0 + dt / 2.0 * B; //new value for temp
		state[idx] = ( temp2 * ( 2.0 - temp ) + dt * A ) / temp;

		__syncthreads();
		//Update channels characteristics
		data[threadIdx.x].x = temp2;
		if (power[idx] > 1)
		{
			data[threadIdx.x].x *= temp2;
			if (power[idx] > 2)
			{
				data[threadIdx.x].x *= temp2;
				if (power[idx] > 3)
				{
					data[threadIdx.x].x *= temp2;
					if (power[idx] > 4)
					{
						data[threadIdx.x].x = pow( temp2, power[idx]);
					}
				}
			}
		}
		__syncthreads();

		if((fi & 0x01) && (threadIdx.x != 0)) //TODO: Find a good solution
		{
			data[threadIdx.x-1].x *= data[threadIdx.x].x;
			data[threadIdx.x].x = 0;
		}
		__syncthreads();
		data[threadIdx.x].x = gbar[idx] *data[threadIdx.x].x;
		data[threadIdx.x].y = ek[idx] *data[threadIdx.x].x;
		if(!(fi & 0x01))
			current[channelIndex[idx]].x = data[threadIdx.x].x;
		for (int bit = 2; bit < 5; ++bit) {
			fi = fi >> 1;
			if(fi & 0x01) //FIND it and write it back to Component solver
			{

			}
		}


	}
}

double SolverGates::Input()
{
	return 0;
}

double SolverGates::Process()
{
	clock_t	start_time = clock();
	if(_m_statistic.nGates > 0)
	{
		int smem_size = (sizeof(TYPE2_) * _threads.x);
		advanceGates <<<_blocks, _threads, smem_size, _stream>>> (
				_state.device,
				_gk.device,
				_ch_currents_gk_ek.device,
				_power.device,
				_params.device,
				_params_div_min_max.device,
				_gbar.device,
				_ek.device,
				_comptIndex.device,
				_channelIndex.device,
				_gateIndex.device,
				_Vm->device,
				_m_statistic.nGates, _m_statistic.dt);
		assert(hipSuccess == hipGetLastError());
	}

	double elapsed_time = ( std::clock() - start_time );
//	cout << "GATE: " << elapsed_time << endl << flush;
	return elapsed_time;
}

double SolverGates::Output()
{
	clock_t	start_time = clock();

//	_ch_currents_gk_ek.Device2Host_Async(_stream);

	return std::clock() - start_time ;
}

/**
 * Set/Get methods
 */

void SolverGates::SetGateParams(int index, vector<double>& params)
{
	for (int i = 0; i < min((int)params.size(),13); ++i)
		_params[index].p[i] = (TYPE_)params[i];

	_params_div_min_max[index].x = (TYPE_)params[PARAMS_DIV];
	_params_div_min_max[index].y = (TYPE_)params[PARAMS_MIN];
	_params_div_min_max[index].z = (TYPE_)params[PARAMS_MAX];
}

void SolverGates::SetValue(int index, FIELD::GATE field, TYPE_ value)
{
	switch(field)
	{
		case FIELD::GATE_CH_GBAR:
			_gbar[index] = value;
			break;
		case FIELD::GATE_CH_GK:
			_gk[index] = value;
			_ch_currents_gk_ek[_channelIndex[index]].x = value;
			break;
		case FIELD::GATE_CH_EK:
			_ek[index] = value;
			_ch_currents_gk_ek[_channelIndex[index]].y = value;
			break;
		case FIELD::GATE_POWER:
			_power[index] = (unsigned char)value;
			break;
		case FIELD::GATE_STATE:
			_state[index] = value;
			break;
		case FIELD::GATE_COMPONENT_INDEX:
			_comptIndex[index] = (int)value;
			break;
		case FIELD::GATE_CHANNEL_INDEX:
			_channelIndex[index] = (int)value;
			break;
		case FIELD::GATE_INDEX:
			_gateIndex[index] = (int)value;
			break;
	}
}

TYPE_ SolverGates::GetValue(int index, FIELD::GATE field)
{
//	switch(field)
//	{
//		case FIELD::CH_GBAR:
//			return _gbar[index];
//		case FIELD::CH_X_POWER:
//			return _xPower[index];
//		case FIELD::CH_Y_POWER:
//			return _yPower[index];
//		case FIELD::CH_Z_POWER:
//			return _zPower[index];
//	}
	return 0;
}
