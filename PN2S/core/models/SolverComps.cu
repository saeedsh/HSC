#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////
//  SolverComps.cpp
//  Implementation of the Class SolverComps
//  Created on:      27-Dec-2013 7:57:50 PM
//  Original author: Saeed Shariati
///////////////////////////////////////////////////////////

#include "SolverComps.h"
#include "solve.h"

#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

using namespace pn2s::models;
//CuBLAS variables
//hipblasHandle_t _handle;

SolverComps::SolverComps(): _stream(0)
{
}

SolverComps::~SolverComps()
{
}


Error_PN2S SolverComps::AllocateMemory(models::ModelStatistic& s, hipStream_t stream)
{
	_statistic = s;
	_stream = stream;

	if(_statistic.nCompts_per_model == 0)
		return Error_PN2S::NO_ERROR;

	size_t modelSize = s.nCompts_per_model*s.nCompts_per_model;
	size_t vectorSize = s.nModels * s.nCompts_per_model;

	_hm.AllocateMemory(modelSize*s.nModels);
	_rhs.AllocateMemory(vectorSize);
	_Vm.AllocateMemory(vectorSize);
	_Constant.AllocateMemory(vectorSize);
	_VMid.AllocateMemory(vectorSize);
	_Ra.AllocateMemory(vectorSize);
	_CmByDt.AllocateMemory(vectorSize);
	_EmByRm.AllocateMemory(vectorSize);
	_InjectBasal.AllocateMemory(vectorSize);
	_InjectVarying.AllocateMemory(vectorSize, 0);

	//Connection to Channels
	_channelIndex.AllocateMemory(vectorSize*2, 0); //Filled with zero

	return Error_PN2S::NO_ERROR;
}

void SolverComps::PrepareSolver(PField<ChannelCurrent, ARCH_>*  channels_current, PField<TYPE_, ARCH_> * Vchannel)
{
	if(_statistic.nCompts_per_model == 0)
		return;

	_channels_current = channels_current;
	_channels_voltage = Vchannel;

	//Copy to GPU
	_hm.Host2Device_Async(_stream);
	_EmByRm.Host2Device_Async(_stream);
	_CmByDt.Host2Device_Async(_stream);
	_channelIndex.Host2Device_Async(_stream);
	_Vm.Host2Device_Async(_stream);
	_InjectBasal.Host2Device_Async(_stream);
	_Constant.Host2Device_Async(_stream);

//	//Create Cublas
//	if ( hipblasCreate(&_handle) != HIPBLAS_STATUS_SUCCESS)
//	{
//		return Error_PN2S(Error_PN2S::CuBLASError,
//				"CUBLAS initialization failed");
//	}
}

/**
 * 			UPDATE MATRIX
 *
 * RHS = Vm * Cm / ( dt / 2.0 ) + Em/Rm;
 *
 */

__global__ void update_rhs(
		TYPE_* hm,
		TYPE_* rhs,
		TYPE_* vm,
		TYPE_* constants,
		size_t nCompt,
		TYPE_* cmByDt,
		TYPE_* emByRm,
		TYPE_* inject_basal,
		TYPE_* inject_varying,
		int* channelIndex,
		ChannelCurrent* channels_current,
		unsigned int size,
		TYPE_ dt)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size){ //For each compartment

    	TYPE_ GkSum   = 0.0;
    	TYPE_ GkEkSum = 0.0;
    	if(channelIndex[idx << 1])
    	{
    		size_t pos = channelIndex[idx << 1 | 0x01];
    		for ( int i = 0; i < channelIndex[idx << 1]; ++i)
			{
				GkSum   += channels_current[pos+i]._gk;
				GkEkSum += channels_current[pos+i]._gk * channels_current[pos+i]._ek;
			}
    	}
//    	// diagonal (a) = below (c) + GkSum
//
//    	unsigned int pos_c = pos_a - !!(pos_localIndex);//C is one back of the A,
//    	    											//if A is the first item, then C is same as A
//    	    											//With this trick we eliminate wrap division
    	//Find location of A and C in the matrix
		unsigned int pos_matrix = (unsigned int)(idx / nCompt) * nCompt * nCompt;
		unsigned int pos_localIndex = idx % nCompt;

		unsigned int pos_a = pos_matrix + pos_localIndex * (nCompt+1) ;

    	hm[pos_a] = constants[idx] + GkSum; //TODO: Check this value for more complex models
    	rhs[idx] = vm[idx] * cmByDt[idx] + emByRm[idx] + GkEkSum;

    	//Injects from basal or varying resources
    	rhs[idx] += inject_basal[idx] + inject_varying[idx];
    }
}


__global__ void update_vm(TYPE_* vm, TYPE_* vmid, int* channelIndex, TYPE_* channels_voltage, size_t size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size){
    	vm[idx] = 2.0 * vmid[idx]- vm[idx];
    	if(channelIndex[idx << 1])
		{
			size_t pos = channelIndex[idx << 1 | 0x01];
			for ( int i = 0; i < channelIndex[idx << 1]; ++i)
				channels_voltage[pos+i] = vm[idx];
		}
    }
}

void SolverComps::Input()
{
	_InjectVarying.Host2Device();
}

void SolverComps::Process()
{
	uint vectorSize = _statistic.nModels * _statistic.nCompts_per_model;

	dim3 threads, blocks;
	threads=dim3(min((vectorSize&0xFFFFFFC0)|0x20,256), 1); //TODO: Check
	blocks=dim3(max(vectorSize / threads.x,1), 1);

//	_hm.print();
//	_rhs.print();
//	_Constant.print();
//	_channels_current->Device2Host();
//	_channels_current->print();

	update_rhs <<<blocks, threads,0, _stream>>> (
			_hm.device,
			_rhs.device,
			_Vm.device,
			_Constant.device,
			_statistic.nCompts_per_model,
			_CmByDt.device,
			_EmByRm.device,
			_InjectBasal.device,
			_InjectVarying.device,
			_channelIndex.device,
			_channels_current->device,
			vectorSize,
			_statistic.dt);
	assert(hipSuccess == hipGetLastError());

	hipStreamSynchronize(_stream);

//	_hm.Device2Host();
//	_hm.print();
//	_rhs.Device2Host();
//	_rhs.print();
	assert(!dsolve_batch (_hm.device, _rhs.device, _VMid.device, _statistic.nCompts_per_model, _statistic.nModels, _stream));

//	_Vm.Device2Host();
//	_Vm.print();
//	_VMid.Device2Host();
//	_VMid.print();

	update_vm <<<blocks, threads,0, _stream>>> (
				_Vm.device,
				_VMid.device,
				_channelIndex.device,
				_channels_voltage->device,
				vectorSize);

//	_Vm.Device2Host_Async(_stream);
//	_Vm.print();
//	_VMid.Device2Host();
//	_VMid.print();


	assert(hipSuccess == hipGetLastError());
//	hipStreamSynchronize(_stream);
}


void SolverComps::Output()
{
	_Vm.Device2Host_Async(_stream);
	hipStreamSynchronize(_stream);
}

void SolverComps::SetValue(int index, FIELD::TYPE field, TYPE_ value)
{
	switch(field)
	{
		case FIELD::CM_BY_DT:
			_CmByDt[index] = value;
			break;
		case FIELD::EM_BY_RM:
			_EmByRm[index] = value;
			break;
		case FIELD::RA:
			_Ra[index] = value;
			break;
		case FIELD::VM:
			_Vm[index] = value;
			break;
		case FIELD::INIT_VM:
			_Vm[index] = value;
			break;
		case FIELD::INJECT_BASAL:
			_InjectBasal[index] = value;
			break;
		case FIELD::INJECT_VARYING:
			_InjectVarying[index] = value;
			break;
		case FIELD::CONSTANT:
			_Constant[index] = value;
			break;
	}
}

TYPE_ SolverComps::GetValue(int index, FIELD::TYPE field)
{
	switch(field)
	{
		case FIELD::CM_BY_DT:
			return _CmByDt[index];
		case FIELD::EM_BY_RM:
			return _EmByRm[index];
		case FIELD::RA:
			return _Ra[index];
		case FIELD::VM:
			return _Vm[index];
		case FIELD::INIT_VM:
			return _Vm[index];
	}
}

void SolverComps::SetHinesMatrix(int n, int row, int col, TYPE_ value)
{
	_hm[_statistic.nCompts_per_model*_statistic.nCompts_per_model*n + row *_statistic.nCompts_per_model + col] = value;
}

void SolverComps::ConnectChannel(int cmpt_index, int ch_index)
{
	if (_channelIndex[cmpt_index*2] == 0)
		_channelIndex[cmpt_index*2+1] = ch_index;
	_channelIndex[cmpt_index*2]++;
}
