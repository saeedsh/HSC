#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////
//  SolverComps.cpp
//  Implementation of the Class SolverComps
//  Created on:      27-Dec-2013 7:57:50 PM
//  Original author: Saeed Shariati
///////////////////////////////////////////////////////////

#include "SolverComps.h"
#include "solve.h"

#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

using namespace pn2s::models;
//CuBLAS variables
//hipblasHandle_t _handle;

hipStream_t _stream;

SolverComps::SolverComps(): _stream(0)
{
}

SolverComps::~SolverComps()
{
}


Error_PN2S SolverComps::AllocateMemory(models::ModelStatistic& s, hipStream_t stream)
{
	_stat = s;
	_stream = stream;

	if(_stat.nCompts == 0)
		return Error_PN2S::NO_ERROR;

	size_t modelSize = s.nCompts*s.nCompts;
	size_t vectorSize = s.nModels * s.nCompts;

	_hm.AllocateMemory(modelSize*s.nModels);
	_rhs.AllocateMemory(vectorSize);
	_Vm.AllocateMemory(vectorSize);
	_VMid.AllocateMemory(vectorSize);
	_Ra.AllocateMemory(vectorSize);
	_CmByDt.AllocateMemory(vectorSize);
	_EmByRm.AllocateMemory(vectorSize);

	_currentIndex.AllocateMemory(vectorSize*2,0);
	_current.AllocateMemory(_stat.nChannels*2);

	return Error_PN2S::NO_ERROR;
}

void SolverComps::PrepareSolver()
{
	if(_stat.nCompts == 0)
		return;

	//Copy to GPU
	_hm.Host2Device_Async(_stream);
	_EmByRm.Host2Device_Async(_stream);

//	//Create Cublas
//	if ( hipblasCreate(&_handle) != HIPBLAS_STATUS_SUCCESS)
//	{
//		return Error_PN2S(Error_PN2S::CuBLASError,
//				"CUBLAS initialization failed");
//	}
}

/**
 * 			UPDATE MATRIX
 *
 * RHS = Vm * Cm / ( dt / 2.0 ) + Em/Rm;
 *
 */

__global__ void update_rhs(TYPE_* rhs, TYPE_* vm, TYPE_* cmByDt, TYPE_* emByRm, size_t size, TYPE_ dt)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size){

    	rhs[idx] = vm[idx] * cmByDt[idx] + emByRm[idx];
    }
}


__global__ void update_vm(TYPE_* vm, TYPE_* vmid, size_t size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    	vm[idx] = 2.0 * vmid[idx]- vm[idx];
}

void SolverComps::Input()
{
//	//Copy to GPU
//	_rhs.Send2Device_Async(_Em,_stream); // Em -> rhs
//	_Rm.Host2Device_Async(_stream);
//	_Vm.Host2Device_Async(_stream);
//	_Cm.Host2Device_Async(_stream);
}

void SolverComps::Process()
{
	uint vectorSize = _stat.nModels * _stat.nCompts;

	dim3 threads, blocks;
	threads=dim3(min((vectorSize&0xFFFFFFC0)|0x20,256), 1); //TODO: Check
	blocks=dim3(max(vectorSize / threads.x,1), 1);

	update_rhs <<<blocks, threads,0, _stream>>> (
			_rhs.device,
			_Vm.device,
			_CmByDt.device,
			_EmByRm.device,
			vectorSize,
			_stat.dt);
	assert(hipSuccess == hipGetLastError());

//	hipStreamSynchronize(_stream);

	_hm.Device2Host();
	_hm.print();
	_rhs.Device2Host();
	_rhs.print();
	assert(!dsolve_batch (_hm.device, _rhs.device, _VMid.device, _stat.nCompts, _stat.nModels, _stream));

	update_vm <<<blocks, threads,0, _stream>>> (
				_Vm.device,
				_VMid.device,
				vectorSize);

	assert(hipSuccess == hipGetLastError());
//	hipStreamSynchronize(_stream);
}


void SolverComps::Output()
{
	_Vm.Device2Host_Async(_stream);
	hipStreamSynchronize(_stream);
}

void SolverComps::SetValue(int index, FIELD::TYPE field, TYPE_ value)
{
	switch(field)
	{
		case FIELD::CM_BY_DT:
			_CmByDt[index] = value;
			break;
		case FIELD::EM_BY_RM:
			_EmByRm[index] = value;
			break;
		case FIELD::RA:
			_Ra[index] = value;
			break;
		case FIELD::VM:
			_Vm[index] = value;
			break;
		case FIELD::INIT_VM:
			_Vm[index] = value;
			break;
	}
}

TYPE_ SolverComps::GetValue(int index, FIELD::TYPE field)
{
	switch(field)
	{
		case FIELD::CM_BY_DT:
			return _CmByDt[index];
		case FIELD::EM_BY_RM:
			return _EmByRm[index];
		case FIELD::RA:
			return _Ra[index];
		case FIELD::VM:
			return _Vm[index];
		case FIELD::INIT_VM:
			return _Vm[index];
	}
}

void SolverComps::SetA(int index, int row, int col, TYPE_ value)
{
	_hm[_stat.nCompts*_stat.nCompts*index + row *_stat.nCompts + col] = value;
}

void SolverComps::AddChannelCurrent(int index, TYPE_ gk, TYPE_ ek)
{
	_currentIndex[index*2]++; //Number of Channels
	if (_currentIndex[index*2+1] == 0)
		_currentIndex[index*2+1] = _current.extra;

	_current[_current.extra++] = gk;
	_current[_current.extra++] = ek;
}
